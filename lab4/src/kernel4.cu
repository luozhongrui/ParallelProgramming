#include <cstddef>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int resX, int maxIterations, int *output, int tile) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int tx = (threadIdx.x + blockIdx.x * blockDim.x) * tile;
    int ty = (threadIdx.y + blockIdx.y * blockDim.y) * tile;
    for(int rowIdx = 0; rowIdx < tile; rowIdx++){
        for(int colIdx = 0; colIdx < tile; colIdx++){
            float x = lowerX + (tx + colIdx) * stepX;
            float y = lowerY + (ty + rowIdx) * stepY;

            float z_re = x;
            float z_im = y;
            int i;
            for(i = 0; i< maxIterations; ++i){
                if(z_re * z_re + z_im * z_im > 4.f)
                    break;
                float new_x = z_re * z_re - z_im * z_im;
                float new_y = 2.f * z_re * z_im;
                z_re = x + new_x;
                z_im = y + new_y;
            }
            int *row = (int *) ((char *)output + (ty + rowIdx) * resX);
            row[tx + colIdx] = i;
        }
    }

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    int dataSize = resX * resY * sizeof(int);
    int *hostData = NULL;
    int *deviceData = NULL;
    size_t pitch;
    int tile = 2;
    hipHostAlloc(&hostData, dataSize, hipHostMallocMapped);
    hipMallocPitch(&deviceData, &pitch, resX * sizeof(int), resY);
    dim3 threadPerBlock(32, 25);
    dim3 blockPerGrid(resX / threadPerBlock.x / tile, resY / threadPerBlock.y / tile);
   mandelKernel<<<blockPerGrid, threadPerBlock>>>(lowerX,  lowerY,  stepX,  stepY,  pitch, maxIterations, deviceData, tile);
   hipMemcpy2D(hostData, resX * sizeof(int), deviceData, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
   memcpy(img, hostData, dataSize);
   hipFree(deviceData);
   hipHostFree(hostData);
}
